#include "hip/hip_runtime.h"
#include <CUDA/_CUDA_NBody_Common.h>
#include <cstdlib>
#include <new>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <>
#include <hip/device_functions.h>
#include <CUDA/hip/hip_vector_types.h>

#define __launch_bounds__(...) \
        __annotate__(launch_bounds(__VA_ARGS__))

#define dt 0.0005f

//__constant__ float dt;
//__constant__ float G;
//__constant__ unsigned int num;
__global__ __launch_bounds__(1024) void positionCalc(NBodyCUDAParticle* particles)
{
	unsigned int id = threadIdx.x + blockIdx.x * 1024;
	particles[id].position += particles[id].velocity * dt;
}
//__global__ void velocityCalc(NBodyCUDAParticle* particles)
//{
//	unsigned int id = threadIdx.x + blockIdx.x * 1024;
//	unsigned int c0 = 0;
//	float3 r = particles[id].position;
//	float3 dv = make_float3(0);
//	for (; c0 < 20 * 1024; ++c0)
//	{
//		float3 dr = particles[c0].position - r;
//		dv += (particles[c0].mass / (powf(dot(dr, dr), 1.5) + 0.00001)) * dr;
//	}
//	particles[id].velocity += dv * 0.001f * 0.005f;
//}
struct PosM
{
	float3 position;
	float mass;
};
__global__ void velocityCalc_Optimize1(NBodyCUDAParticle* particles)
{
	__shared__ PosM posM[1024];
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	float3 r = particles[id].position;
	float3 dv = make_float3(0);
	for (int c0(0); c0 < gridDim.x; ++c0)
	{
		posM[threadIdx.x] = *(PosM*)(particles + threadIdx.x + c0 * blockDim.x);
		__syncthreads();
		for (int c1(0); c1 < blockDim.x; ++c1)
		{
			float3 dr = posM[c1].position - r;
			float drr = rsqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + 0.00001f);
			// float drr = rsqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + 0.00000001f);
			drr = drr * drr * drr;
			dv += (posM[c1].mass * drr) * dr;
		}
		__syncthreads();
	}
	particles[id].velocity += dv * 0.001f * dt;
}
__global__ void forceCalc(NBodyCUDAParticle* particles, ExpData* expData)
{
	__shared__ PosM posM[1024];
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	float3 r = particles[id].position;
	float3 force = make_float3(0);
	for (int c0(0); c0 < gridDim.x; ++c0)
	{
		posM[threadIdx.x] = *(PosM*)(particles + threadIdx.x + c0 * blockDim.x);
		__syncthreads();
		for (int c1(0); c1 < blockDim.x; ++c1)
		{
			float3 dr = posM[c1].position - r;
			float drr = rsqrtf(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + 0.001f);
			drr = drr * drr * drr;
			force += (posM[c1].mass * drr) * dr;
		}
		__syncthreads();
	}
	float a(sqrtf(r.x * r.x + r.y * r.y + r.z * r.z));
	float fr = dot(force, r) / a;
	expData[id] = { a,fr };
}
#undef dt
NBodyCUDA_Glue::NBodyCUDA_Glue(unsigned int _blocks, float _dt, float _G)
	:
	blocks(_blocks),
	stream(nullptr)
{
	unsigned int _num(1024 * _blocks);
	hipStreamCreate(&stream);
	//hipMemcpyToSymbol(HIP_SYMBOL(&dt), &_dt, sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(&G), &_G, sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(&num), &_num, sizeof(unsigned int));
}
void NBodyCUDA_Glue::run()
{
	positionCalc << < dim3(blocks, 1, 1), dim3(1024, 1, 1), 0, stream >> > (particles);
	velocityCalc_Optimize1 << < dim3(blocks, 1, 1), dim3(1024, 1, 1), 0, stream >> > (particles);
	hipStreamSynchronize(stream);
}
void NBodyCUDA_Glue::experiment(ExpData* expData)
{
	forceCalc << < dim3(blocks, 1, 1), dim3(1024, 1, 1), 0, stream >> > (particles, expData);
	hipStreamSynchronize(stream);
}