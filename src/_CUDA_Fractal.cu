#include "hip/hip_runtime.h"
#include <CUDA/_CUDA_Fractal.h>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <>
#include <hip/device_functions.h>
#include <cuda_gl_interop.h>
#include <CUDA/hip/hip_vector_types.h>

// detailed impl of glue

// float version
__device__ float mandelbrotFractalKernel(float2 c, int iter)
{
	float c2 = dot(c, c);
	// skip computation inside M1 - https://iquilezles.org/articles/mset1bulb
	if (256.f * c2 * c2 - 96.0f * c2 + 32.0f * c.x - 3.0f < 0.0f) return 0.f;
	// skip computation inside M2 - https://iquilezles.org/articles/mset2bulb
	if (16.f * (c2 + 2.f * c.x + 1.f) - 1.f < 0.f) return 0.f;

	float B = iter / 2;//256.f;
	float l = 0.f;
	float2 z = make_float2(0.f);
	for (int i = 0; i < iter; i++)
	{
		z = make_float2(z.x * z.x - z.y * z.y, 2.f * z.x * z.y) + c;
		if (dot(z, z) > (B * B)) break;
		l += 1.f;
	}
	if (l > iter - 1) return 0.f;

	float sl = l - __log2f(__log2f(dot(z, z))) + 4.f;
	return sl;
}

// double version
__device__ double mandelbrotFractalKernel(double2 c, int iter)
{
	double c2 = dot(c, c);
	// skip computation inside M1 - https://iquilezles.org/articles/mset1bulb
	if (256. * c2 * c2 - 96. * c2 + 32. * c.x - 3. < 0.) return 0.;
	// skip computation inside M2 - https://iquilezles.org/articles/mset2bulb
	if (16. * (c2 + 2. * c.x + 1.) - 1. < 0.) return 0.;

	int B = iter / 2;//256.f;
	double l = 0.;
	double2 z = make_double2(0.);
	for (int i = 0; i < iter; i++)
	{
		z = make_double2(z.x * z.x - z.y * z.y, 2. * z.x * z.y) + c;
		if (dot(z, z) > (B * B)) break;
		l += 1.;
	}
	if (l > iter - 1) return 0.;

	double sl = l - std::log(std::log(dot(z, z))) + 4.;
	return sl;
}

// 32 * 32 block
__global__ void runMandelbrotFractal(hipSurfaceObject_t img, int2 _size, double2 _center, double scale, int iter, bool useDouble = false)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < _size.x && y < _size.y)
	{
		float3 col = make_float3(0.f);
#define AA 2
		for (int m = 0; m < AA; m++)
			for (int n = 0; n < AA; n++)
			{
				if (useDouble)
				{
					double2 p = _center + make_double2(x - _size.x / 2 + m / double(AA), _size.y / 2 - y + n / double(AA)) / (double(_size.x) * scale);
					double l = 3. + mandelbrotFractalKernel(p, iter) * 0.15;
					// col += 0.5f * (1.f + make_float3(std::cos(l + 0.f), std::cos(l + 0.6f), std::cos(l + 1.f)));
					col += make_float3(0.5 * (1. + make_double3(cos(l + 0.), cos(l + 0.6), cos(l + 1.))));
				}
				else
				{
					float2 p = make_float2(_center) + make_float2(x - _size.x / 2 + m / float(AA), _size.y / 2 - y + n / float(AA)) / (float(_size.x) * scale);
					float l = 3.f + mandelbrotFractalKernel(p, iter) * 0.15f;
					col += 0.5f * (1.f + make_float3(__cosf(l + 0.f), __cosf(l + 0.6f), __cosf(l + 1.f)));
				}
			}
		col /= float(AA * AA);
		surf2Dwrite(make_float4(col, 1.f), img, x * sizeof(float4), y);
	}
}

MandelbrotFractalCUDA_Glue::MandelbrotFractalCUDA_Glue(OpenGL::MandelbrotFractalData* _fractalData, OpenGL::TextureConfig<OpenGL::TextureStorage2D>* _textureConfig)
	:
	fractalData(_fractalData)
{
	hipStreamCreate(&stream);
	if (_textureConfig)
	{
		resize(_textureConfig);
	}
}

MandelbrotFractalCUDA_Glue::~MandelbrotFractalCUDA_Glue()
{
	close();
	hipStreamDestroy(stream);
}

void MandelbrotFractalCUDA_Glue::resize(OpenGL::TextureConfig<OpenGL::TextureStorage2D>* _textureConfig)
{
	img.registerImage(*_textureConfig, hipGraphicsRegisterFlagsSurfaceLoadStore);
}

void MandelbrotFractalCUDA_Glue::run()
{
	hipStreamSynchronize(stream);
	img.map(stream);
	img.createArray(0);
	img.createSurface();

	dim3 grid = { unsigned((fractalData->size.w + 31) / 32), unsigned((fractalData->size.h + 31) / 32), 1 };
	int2 size = make_int2(fractalData->size.w, fractalData->size.h);
	double2 center = make_double2(fractalData->center[0], fractalData->center[1]);
	runMandelbrotFractal << < grid, { 32, 32, 1 }, 0, stream >> > (img.surface, size, center, fractalData->scale, fractalData->iter, fractalData->useDouble);

	img.destroySurface();
	img.unmap(stream);
	hipStreamSynchronize(stream);
}

void MandelbrotFractalCUDA_Glue::close()
{
	img.unregisterResource();
}